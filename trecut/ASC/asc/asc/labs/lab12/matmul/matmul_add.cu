#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void MatAdd(float* A, float* B, float* C)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;
 
if (i < N && j < N)
C[i*N+j] = A[i*N+j] + B[i*N+j];
}
 
int main()
{
 
float a[N*N], b[N*N], c[N*N]; // host matrices
float *A, *B, *C;	// device matrices
	int i,j;
	for (i = 0; i < N; i++ )
	hipMalloc( (void**) &A, N *N); 
 	hipMemcpy( A, a, N*N, hipMemcpyHostToDevice);

	hipMalloc( (void**) &B, N *N); 
        hipMemcpy( B, b, N*N, hipMemcpyHostToDevice);

	hipMalloc( (void**) &C, N * N)); 

// Kernel invocation
dim3 dimBlock(BLK, BLK);
dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x, (N + dimBlock.y - 1) / dimBlock.y);
MatAdd<<<dimGrid, dimBlock>>>(A, B, C);
 hipMemcpy( c, C, N*N, hipMemcpyDeviceToHost);
}
